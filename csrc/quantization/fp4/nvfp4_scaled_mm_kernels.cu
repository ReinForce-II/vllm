/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cutlass_extensions/common.hpp"

#include "cutlass/cutlass.h"

#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/packed_stride.hpp"

using namespace cute;

#define CHECK_TYPE(x, st, m) \
  TORCH_CHECK(x.scalar_type() == st, ": Inconsistency of Tensor type:", m)
#define CHECK_TH_CUDA(x, m) \
  TORCH_CHECK(x.is_cuda(), m, ": must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) \
  TORCH_CHECK(x.is_contiguous(), m, ": must be contiguous")
#define CHECK_INPUT(x, st, m) \
  CHECK_TH_CUDA(x, m);        \
  CHECK_CONTIGUOUS(x, m);     \
  CHECK_TYPE(x, st, m)

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;
constexpr auto SF_DTYPE = at::ScalarType::Float8_e4m3fn;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
// Kernel Perf config
template <typename T>
struct KernelTraitsSm100;

template <>
struct KernelTraitsSm100<float> {
  using MmaTileShape = Shape<_128, _128, _256>;
  using ClusterShape = Shape<_1, _1, _1>;
  using PerSmTileShape_MNK = Shape<_128, _128, _256>;
};

template <>
struct KernelTraitsSm100<cutlass::half_t> {
  using MmaTileShape = Shape<_256, _256, _256>;
  using ClusterShape = Shape<_4, _4, _1>;
  using PerSmTileShape_MNK = Shape<_128, _256, _256>;
};

template <>
struct KernelTraitsSm100<cutlass::bfloat16_t> {
  using MmaTileShape = Shape<_256, _256, _256>;
  using ClusterShape = Shape<_4, _4, _1>;
  using PerSmTileShape_MNK = Shape<_128, _256, _256>;
};

template <typename T>
struct Fp4GemmSm100 {
  // A matrix configuration
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  // B matrix configuration
  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  // C/D matrix configuration
  using ElementD = T;
  using ElementC = T;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  // Kernel functional config
  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm100;
  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  // Kernel Perf config
  using MmaTileShape = typename KernelTraitsSm100<T>::MmaTileShape;
  using ClusterShape = typename KernelTraitsSm100<T>::ClusterShape;
  using PerSmTileShape_MNK = typename KernelTraitsSm100<T>::PerSmTileShape_MNK;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, PerSmTileShape_MNK, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, MmaTileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using StrideA = typename Gemm::GemmKernel::StrideA;
  using LayoutA = decltype(cute::make_layout(make_shape(0, 0, 0), StrideA{}));
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using LayoutB = decltype(cute::make_layout(make_shape(0, 0, 0), StrideB{}));
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using LayoutC = decltype(cute::make_layout(make_shape(0, 0, 0), StrideC{}));
  using StrideD = typename Gemm::GemmKernel::StrideD;
  using LayoutD = decltype(cute::make_layout(make_shape(0, 0, 0), StrideD{}));
};

template <typename T>
typename T::Gemm::Arguments args_from_options_sm100(
    at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
    at::Tensor const& A_sf, at::Tensor const& B_sf, at::Tensor const& alpha,
    int64_t M, int64_t N, int64_t K) {
  using ElementA = typename T::Gemm::ElementA;
  using ElementB = typename T::Gemm::ElementB;
  using ElementSFA = cutlass::float_ue4m3_t;
  using ElementSFB = cutlass::float_ue4m3_t;
  using ElementD = typename T::Gemm::ElementD;
  using ElementCompute = float;
  using StrideA = typename T::StrideA;
  using StrideB = typename T::StrideB;
  using StrideD = typename T::StrideD;
  using Sm100BlkScaledConfig =
      typename T::Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  int m = static_cast<int>(M);
  int n = static_cast<int>(N);
  int k = static_cast<int>(K);
  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {m, k, 1});
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {n, k, 1});
  auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {m, n, 1});

  auto layout_SFA = Sm100BlkScaledConfig::tile_atom_to_shape_SFA(
      cute::make_shape(m, n, k, 1));
  auto layout_SFB = Sm100BlkScaledConfig::tile_atom_to_shape_SFB(
      cute::make_shape(m, n, k, 1));

  typename T::Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      {// Mainloop arguments
       static_cast<ElementA const*>(A.data_ptr()), stride_A,
       static_cast<ElementB const*>(B.data_ptr()), stride_B,
       static_cast<ElementSFA const*>(A_sf.data_ptr()), layout_SFA,
       static_cast<ElementSFB const*>(B_sf.data_ptr()), layout_SFB},
      {     // Epilogue arguments
       {},  // epilogue.thread
       static_cast<ElementD const*>(D.data_ptr()),
       stride_D,
       static_cast<ElementD*>(D.data_ptr()),
       stride_D}};
  auto& fusion_args = arguments.epilogue.thread;
  fusion_args.alpha_ptr = static_cast<ElementCompute const*>(alpha.data_ptr());
  return arguments;
}

template <typename T>
void runGemmSm100(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
                  at::Tensor const& A_sf, at::Tensor const& B_sf,
                  at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
                  hipStream_t stream) {
  typename Fp4GemmSm100<T>::Gemm gemm;

  auto arguments = args_from_options_sm100<Fp4GemmSm100<T>>(D, A, B, A_sf, B_sf,
                                                            alpha, m, n, k);

  size_t workspace_size = Fp4GemmSm100<T>::Gemm::get_workspace_size(arguments);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(A.device());
  auto workspace = torch::empty(workspace_size, workspace_options);

  CUTLASS_CHECK(gemm.can_implement(arguments));

  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));

  CUTLASS_CHECK(gemm.run(arguments, workspace.data_ptr(), stream));
}
#else
template <typename T>
void runGemmSm100(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
                  at::Tensor const& A_sf, at::Tensor const& B_sf,
                  at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
                  hipStream_t stream) {
  TORCH_CHECK(false,
              "Unsupported CUTLASS version. Set VLLM_CUTLASS_SRC_DIR to "
              "a CUTLASS 3.8 source directory to enable support.");
}
#endif  // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

#if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)
// ** not using template specialization, some unexpected issues here
struct Fp4GemmSm120Float16 {
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  using ElementC = cutlass::half_t;
  using ElementD = cutlass::half_t;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm120;

  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  using ThreadBlockShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_1, _1, _1>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ThreadBlockShape, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, ThreadBlockShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using LayoutA = decltype(cute::make_layout(make_shape(0, 0, 0), StrideA{}));
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using LayoutB = decltype(cute::make_layout(make_shape(0, 0, 0), StrideB{}));
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using LayoutC = decltype(cute::make_layout(make_shape(0, 0, 0), StrideC{}));
  using StrideD = typename Gemm::GemmKernel::StrideD;
  using LayoutD = decltype(cute::make_layout(make_shape(0, 0, 0), StrideD{}));
};

struct Fp4GemmSm120Bfloat16 {
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  using ElementC = cutlass::bfloat16_t;
  using ElementD = cutlass::bfloat16_t;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm120;

  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  using ThreadBlockShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_1, _1, _1>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ThreadBlockShape, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, ThreadBlockShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using LayoutA = decltype(cute::make_layout(make_shape(0, 0, 0), StrideA{}));
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using LayoutB = decltype(cute::make_layout(make_shape(0, 0, 0), StrideB{}));
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using LayoutC = decltype(cute::make_layout(make_shape(0, 0, 0), StrideC{}));
  using StrideD = typename Gemm::GemmKernel::StrideD;
  using LayoutD = decltype(cute::make_layout(make_shape(0, 0, 0), StrideD{}));
};

struct Fp4GemmSm120Float32 {
  using ElementA = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutATag = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 32;

  using ElementB = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
  using LayoutBTag = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 32;

  using ElementC = float;
  using ElementD = float;
  using LayoutCTag = cutlass::layout::RowMajor;
  using LayoutDTag = cutlass::layout::RowMajor;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  using ElementAccumulator = float;
  using ArchTag = cutlass::arch::Sm120;

  using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

  using ThreadBlockShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_1, _1, _1>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ThreadBlockShape, ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto, ElementAccumulator,
          ElementAccumulator, ElementC, LayoutCTag, AlignmentC, ElementD,
          LayoutDTag, AlignmentD,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag, OperatorClass, ElementA, LayoutATag, AlignmentA, ElementB,
          LayoutBTag, AlignmentB, ElementAccumulator, ThreadBlockShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          cutlass::gemm::collective::KernelScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using LayoutA = decltype(cute::make_layout(make_shape(0, 0, 0), StrideA{}));
  using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using LayoutB = decltype(cute::make_layout(make_shape(0, 0, 0), StrideB{}));
  using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using LayoutC = decltype(cute::make_layout(make_shape(0, 0, 0), StrideC{}));
  using StrideD = typename Gemm::GemmKernel::StrideD;
  using LayoutD = decltype(cute::make_layout(make_shape(0, 0, 0), StrideD{}));
};

template <typename T>
auto make_args(void const* A, void const* B, void const* C, void* D,
               void const* SFA, void const* SFB, void const* alpha, int M,
               int N, int K) {
  using namespace cute;

  typename T::StrideA stride_A;
  typename T::LayoutA layout_A;
  typename T::LayoutSFA layout_SFA;
  typename T::StrideB stride_B;
  typename T::LayoutB layout_B;
  typename T::LayoutSFB layout_SFB;
  typename T::StrideC stride_C;
  typename T::LayoutC layout_C;
  typename T::StrideD stride_D;
  typename T::LayoutD layout_D;

  stride_A = cutlass::make_cute_packed_stride(typename T::StrideA{}, {M, K, 1});
  stride_B = cutlass::make_cute_packed_stride(typename T::StrideB{}, {N, K, 1});
  stride_C = cutlass::make_cute_packed_stride(typename T::StrideC{}, {M, N, 1});
  stride_D = cutlass::make_cute_packed_stride(typename T::StrideD{}, {M, N, 1});

  layout_A = make_layout(make_shape(M, K, 1), stride_A);
  layout_B = make_layout(make_shape(N, K, 1), stride_B);
  layout_C = make_layout(make_shape(M, N, 1), stride_C);
  layout_D = make_layout(make_shape(M, N, 1), stride_D);
  layout_SFA = T::Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig::
      tile_atom_to_shape_SFA(cute::make_shape(M, N, K, 1));
  layout_SFB = T::Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig::
      tile_atom_to_shape_SFB(cute::make_shape(M, N, K, 1));

  typename T::Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K, 1},
      {
          // Mainloop arguments
          static_cast<typename T::ElementA::DataType const*>(A),
          stride_A,
          static_cast<typename T::ElementB::DataType const*>(B),
          stride_B,
          static_cast<typename T::ElementA::ScaleFactorType const*>(SFA),
          layout_SFA,
          static_cast<typename T::ElementB::ScaleFactorType const*>(SFB),
          layout_SFB,
      },
      {{},
       static_cast<typename T::ElementC const*>(C),
       stride_C,
       static_cast<typename T::ElementD*>(D),
       stride_D}};
  auto& fusion_args = arguments.epilogue.thread;
  fusion_args.alpha_ptr = static_cast<float const*>(alpha);
  return arguments;
}

template <typename T>
void runGemmSm120(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
                  at::Tensor const& A_sf, at::Tensor const& B_sf,
                  at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
                  hipStream_t stream) {
  using Gemm = typename T::Gemm;
  Gemm gemm;

  auto arguments =
      make_args<T>(A.data_ptr(), B.data_ptr(), A_sf.data_ptr(), D.data_ptr(),
                   A_sf.data_ptr(), B_sf.data_ptr(), alpha.data_ptr(), m, n, k);

  size_t workspace_size = Gemm::get_workspace_size(arguments);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(A.device());
  auto workspace = torch::empty(workspace_size, workspace_options);

  CUTLASS_CHECK(gemm.can_implement(arguments));
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.data_ptr(), stream));
  CUTLASS_CHECK(gemm.run(arguments, workspace.data_ptr(), stream));
}
#else
template <typename T>
void runGemmSm120(at::Tensor& D, at::Tensor const& A, at::Tensor const& B,
                  at::Tensor const& A_sf, at::Tensor const& B_sf,
                  at::Tensor const& alpha, int64_t m, int64_t n, int64_t k,
                  hipStream_t stream) {
  TORCH_CHECK(false,
              "Unsupported CUTLASS version. Set VLLM_CUTLASS_SRC_DIR to "
              "a CUTLASS 3.8 source directory to enable support.");
}
#endif  // defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)

void cutlass_scaled_fp4_mm_sm100a(torch::Tensor& D, torch::Tensor const& A,
                                  torch::Tensor const& B,
                                  torch::Tensor const& A_sf,
                                  torch::Tensor const& B_sf,
                                  torch::Tensor const& alpha) {
  CHECK_INPUT(A, FLOAT4_E2M1X2, "a");
  CHECK_INPUT(B, FLOAT4_E2M1X2, "b");

  CHECK_INPUT(A_sf, SF_DTYPE, "scale_a");
  CHECK_INPUT(B_sf, SF_DTYPE, "scale_b");

  CHECK_INPUT(alpha, at::ScalarType::Float, "alpha");

  TORCH_CHECK(A.dim() == 2, "a must be a matrix");
  TORCH_CHECK(B.dim() == 2, "b must be a matrix");
  TORCH_CHECK(A.sizes()[1] == B.sizes()[1],
              "a and b shapes cannot be multiplied (", A.sizes()[0], "x",
              A.sizes()[1], " and ", B.sizes()[0], "x", B.sizes()[1], ")");

  auto const m = A.sizes()[0];
  auto const n = B.sizes()[0];
  auto const k = A.sizes()[1] * 2;

  constexpr int alignment = 32;
  TORCH_CHECK(k % alignment == 0, "Expected k to be divisible by ", alignment,
              ", but got a shape: (", A.sizes()[0], "x", A.sizes()[1],
              "), k: ", k, ".");
  TORCH_CHECK(n % alignment == 0, "Expected n to be divisible by ", alignment,
              ", but got b shape: (", B.sizes()[0], "x", B.sizes()[1], ").");

  auto round_up = [](int x, int y) { return (x + y - 1) / y * y; };
  int rounded_m = round_up(m, 128);
  int rounded_n = round_up(n, 128);
  // Since k is divisible by 32 (alignment), k / 16 is guaranteed to be an
  // integer.
  int rounded_k = round_up(k / 16, 4);

  TORCH_CHECK(A_sf.dim() == 2, "scale_a must be a matrix");
  TORCH_CHECK(B_sf.dim() == 2, "scale_b must be a matrix");
  TORCH_CHECK(A_sf.sizes()[1] == B_sf.sizes()[1],
              "scale_a and scale_b shapes cannot be multiplied (",
              A_sf.sizes()[0], "x", A_sf.sizes()[1], " and ", B_sf.sizes()[0],
              "x", B_sf.sizes()[1], ")");
  TORCH_CHECK(A_sf.sizes()[0] == rounded_m && A_sf.sizes()[1] == rounded_k,
              "scale_a must be padded and swizzled to a shape (", rounded_m,
              "x", rounded_k, "), but got a shape (", A_sf.sizes()[0], "x",
              A_sf.sizes()[1], ")");
  TORCH_CHECK(B_sf.sizes()[0] == rounded_n && B_sf.sizes()[1] == rounded_k,
              "scale_b must be padded and swizzled to a shape (", rounded_n,
              "x", rounded_k, "), but got a shape (", B_sf.sizes()[0], "x",
              B_sf.sizes()[1], ")");

  auto out_dtype = D.dtype();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(A.get_device());
  const hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();

  if (prop->major == 10) {
    if (out_dtype == at::ScalarType::Half) {
      runGemmSm100<cutlass::half_t>(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                    stream);
    } else if (out_dtype == at::ScalarType::BFloat16) {
      runGemmSm100<cutlass::bfloat16_t>(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                        stream);
    } else if (out_dtype == at::ScalarType::Float) {
      runGemmSm100<float>(D, A, B, A_sf, B_sf, alpha, m, n, k, stream);
    } else {
      TORCH_CHECK(false, "Unsupported output data type of nvfp4 mm");
    }
  } else if (prop->major == 12) {
    if (out_dtype == at::ScalarType::Half) {
      runGemmSm120<Fp4GemmSm120Float16>(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                        stream);
    } else if (out_dtype == at::ScalarType::BFloat16) {
      runGemmSm120<Fp4GemmSm120Bfloat16>(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                         stream);
    } else if (out_dtype == at::ScalarType::Float) {
      runGemmSm120<Fp4GemmSm120Float32>(D, A, B, A_sf, B_sf, alpha, m, n, k,
                                        stream);
    } else {
      TORCH_CHECK(false, "Unsupported output data type of nvfp4 mm");
    }
  } else {
    TORCH_CHECK(false, "Unsupported GPU architecture for nvfp4 mm");
  }
}
